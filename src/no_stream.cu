#include <stdio.h>
#include <hip/hip_runtime.h>

#include <sys/time.h>
#include <unistd.h>
// gettimeofday()

#define MAX_N 102400000

#define VALUE_TYPE int

#define WARP_SIZE 32

#define THREADS_PER_BLOCK 128

#define BENCH_REPEAT 100

__global__ void Kernel(VALUE_TYPE *A)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < MAX_N)
    {
        for (size_t i = 0; i < 3500; i++)
        {
            A[idx] = A[idx] * idx;
        }
        // A[idx] = 1;
    }
}

int main()
{
    struct timeval tv_start, tv_end;

    VALUE_TYPE *A;
    VALUE_TYPE *d_A;

    size_t n = MAX_N;

    dim3 gridDim;
    dim3 blockDim;
    blockDim.x = 128;
    gridDim.x = (n + blockDim.x - 1) / blockDim.x;
    printf("CUDA kernel launch with %d blocks of %d threads\n", gridDim.x, blockDim.x);

    hipHostAlloc(&A, n * sizeof(VALUE_TYPE), hipHostMallocDefault);
    hipMemset(A, 0, n * sizeof(VALUE_TYPE));
    hipMalloc((void **)&d_A, n * sizeof(VALUE_TYPE));

    gettimeofday(&tv_start, NULL);
    for (size_t i = 0; i < BENCH_REPEAT; i++)
    {
        hipMemcpy(d_A, A, n * sizeof(VALUE_TYPE), hipMemcpyHostToDevice);
    }
    gettimeofday(&tv_end, NULL);
    printf("hipMemcpy Time: %fms\n", ((tv_end.tv_sec - tv_start.tv_sec) * 1000.0 + (tv_end.tv_usec - tv_start.tv_usec) / 1000.0) / BENCH_REPEAT);

    gettimeofday(&tv_start, NULL);
    for (size_t i = 0; i < BENCH_REPEAT; i++)
    {
        Kernel<<<gridDim, blockDim>>>(d_A);
    }
    hipDeviceSynchronize();
    gettimeofday(&tv_end, NULL);
    printf("Kernel Time: %fms\n", ((tv_end.tv_sec - tv_start.tv_sec) * 1000.0 + (tv_end.tv_usec - tv_start.tv_usec) / 1000.0) / BENCH_REPEAT);

    gettimeofday(&tv_start, NULL);
    for (size_t i = 0; i < BENCH_REPEAT; i++)
    {
        hipMemcpy(A, d_A, n * sizeof(VALUE_TYPE), hipMemcpyDeviceToHost);
    }
    gettimeofday(&tv_end, NULL);
    printf("hipMemcpy Time: %fms\n", ((tv_end.tv_sec - tv_start.tv_sec) * 1000.0 + (tv_end.tv_usec - tv_start.tv_usec) / 1000.0) / BENCH_REPEAT);

    // for (size_t i = 0; i < MAX_N; i++)
    // {
    //     if (A[i] != 1)
    //     {
    //         printf("error\n");
    //         break;
    //     }
    // }

    // gettimeofday(&tv_start, NULL);
    // for (size_t i = 0; i < BENCH_REPEAT; i++)
    // {
    //     cudaMemcpy(d_A, A, n/8 * sizeof(VALUE_TYPE), cudaMemcpyHostToDevice);
    // }
    // gettimeofday(&tv_end, NULL);
    // printf("buf cudaMemcpy Time: %fms\n", ((tv_end.tv_sec - tv_start.tv_sec) * 1000.0 + (tv_end.tv_usec - tv_start.tv_usec) / 1000.0) / BENCH_REPEAT);
    
    // gettimeofday(&tv_start, NULL);
    // for (size_t i = 0; i < BENCH_REPEAT; i++)
    // {
    //     cudaMemcpy(A, d_A, n/8 * sizeof(VALUE_TYPE), cudaMemcpyDeviceToHost);
    //     cudaMemcpy(A, d_A, n/8 * sizeof(VALUE_TYPE), cudaMemcpyDeviceToHost);
    // }
    // gettimeofday(&tv_end, NULL);
    // printf("buf cudaMemcpy Time: %fms\n", ((tv_end.tv_sec - tv_start.tv_sec) * 1000.0 + (tv_end.tv_usec - tv_start.tv_usec) / 1000.0) / BENCH_REPEAT);


    hipHostFree(A);
    hipFree(d_A);

    return 0;
}